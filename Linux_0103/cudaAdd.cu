
#include <hip/hip_runtime.h>
#include <stdio.h>
/* 덧셈을 위한 커널 */
__global__ void add(int *a, int *b, int *sum) {
 *sum = *a + *b;
}


int main(int argc, char** argv)
{
 int a = 2, b = 4, sum;
 int *dev_a, *dev_b, *dev_sum; /* 변수 a, b, sum을 디바이스로 복사하기 위한 변수 */
 int size = sizeof( int ); /* 정수형을 저장하기 위한 크기 */
 /* 변수 a, b, sum을 위한 디바이스의 공간 확보 */
 hipMalloc( (void**)&dev_a, size );
 hipMalloc( (void**)&dev_b, size );
 hipMalloc( (void**)&dev_sum, size );
  /* 디바이스로 복사 */
 hipMemcpy( dev_a, &a, size, hipMemcpyHostToDevice );
 hipMemcpy( dev_b, &b, size, hipMemcpyHostToDevice );
 /* GPU에서 커널 함수 add() 실행 */
 add<<< 1, 1 >>>( dev_a, dev_b, dev_sum );
 /* GPU에서 계산한 결과값을 호스트의 sum 변수로 복사 */
 hipMemcpy( &sum, dev_sum, size, hipMemcpyDeviceToHost );
 /* 사용이 끝난 메모리 공간 해제 */
 hipFree( dev_a );
 hipFree( dev_b );
 hipFree( dev_sum );
 /* 결과 출력 */
 printf("2 + 4 = %d from CUDA\n", sum);
 return 0;
}



